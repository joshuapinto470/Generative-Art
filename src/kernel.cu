#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>

#define TX 32
#define TY 32
#define MAX_ITER 6

typedef double Float;
typedef float3 vec3;
typedef float2 vec2;

__device__ unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

/*Shift range from (0, 1) to (c, d)*/
__device__ float shiftrange(float c, float d, float t)
{
   return c + (float)(d - c) * t;
}

__device__ float fract(float x){
   return x - floorf(x);
}

__device__ float2 fract2(float2 n) {
   return make_float2(fract(n.x), fract(n.y));
}

__device__ float3 cos3(float3 n) {
   return make_float3(cos(n.x), cos(n.y), cos(n.z));
}

__device__ float3 sin3(float3 n) {
   return make_float3(sin(n.x), sin(n.y), sin(n.z));
}

__device__ float3 palette( float t ) {
    float3 a = make_float3(0.5, 0.5, 0.5);
    float3 b = make_float3(0.5, 0.5, 0.5);
    float3 c = make_float3(1.0, 1.0, 1.0);
    float3 d = make_float3(0.263,0.416,0.557);

    return a + b * cos3( 6.28318 * (c*t + d) );
}

__device__ float length(float2 p) {
   return sqrt(p.x * p.x + p.y * p.y);
}

__global__ void cudaKernel(uchar4 *d_out, int w, int h, float iTime)
{
   const int c = blockIdx.x * blockDim.x + threadIdx.x;
   const int r = blockIdx.y * blockDim.y + threadIdx.y;

   if ((c >= w) || (r >= h))
      return;

   const int i = c + r * w;
   vec2 uv = make_float2((float)((c * 2.0) - w) / (float)h,
                  -1.0 * (float)((r * 2.0) - h) / (float)h);

   float L = length(uv);
   vec3 finalColor = make_float3(0.0f, 0.0f, 0.0f);

   for (int i = 0; i < 4; i++) {
      uv = fract2(uv * 1.5) - 0.5;

      double d = length(uv) * exp(-L);
      vec3 col = palette(L + (i + iTime) * 0.4);
      //col = make_float3(0.0, 1.0, 0.5);
      d = sin(d * 8.0 + iTime) / 8.0;
      d = abs(d);
      d = pow(0.01 / d, 1.2);
      finalColor += col * d;
   }
   
   d_out[i].x = clip(finalColor.x * 255.0);
   d_out[i].y = clip(finalColor.y * 255.0);
   d_out[i].z = clip(finalColor.z * 255.0);
   d_out[i].w = 255;

}

void kernelLauncher(uchar4 *d_out, int w, int h, float iTime)
{
   const dim3 gridSize = dim3((w + TX - 1) / TX, (h + TY - 1) / TY);
   const dim3 blockSize(TX, TY);
   cudaKernel<<<gridSize, blockSize>>>(d_out, w, h, iTime);
}